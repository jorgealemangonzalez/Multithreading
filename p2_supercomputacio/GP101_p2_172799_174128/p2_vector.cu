
#include <hip/hip_runtime.h>
#include "stdio.h"
#define N 514    //Para correr con mas threads de los posibles en un bloque
//#define N  65537
__global__ void add(int *a, int *b, int *c)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x; //El id del thread es el id que tiene ese thread dentro de un bloque
	c[tid]=a[tid]+b[tid];							 //El id del bloque es el id que tiene ese bloque dentro del grid			
}													 //La dimension del bloque es el numero de threads que tiene cada bloque

int main()
{
	int a[N], b[N], c[N];//host 
	int *dev_a, *dev_b, *dev_c;//device

	hipMalloc((void**)&dev_a, N*sizeof(int) );  
	hipMalloc((void**)&dev_b, N*sizeof(int) );
	hipMalloc((void**)&dev_c, N*sizeof(int) );


	for (int i = 0; i < N; i++){
		a[i] = i,
		b[i] = 1;
	}

	hipMemcpy(dev_a, a, N*sizeof(int), hipMemcpyHostToDevice); //host to device
	hipMemcpy(dev_b, b, N*sizeof(int), hipMemcpyHostToDevice);

	//add<<<1,N>>>(dev_a,dev_b,dev_c); //Ejecuta 1 bloque con N threads
	add<<<N,1>>>(dev_a,dev_b,dev_c); //Ejecuta N bloques de 1 solo thread cada uno
	
	//Call CUDA kernel
	hipMemcpy(c, dev_c, N*sizeof(int), hipMemcpyDeviceToHost);//Copy memory from device to host
	//copy array to host
	for (int i = 0; i < N; i++)
		printf("%d + %d = %d\n", a[i], b[i], c[i]);
	
	hipFree(dev_a);//free device mem
	hipFree(dev_b);
	hipFree(dev_c);

	return 0;

}

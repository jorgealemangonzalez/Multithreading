
#include <hip/hip_runtime.h>
#include <stdio.h> 
#include <stdlib.h> 

__global__ void fill_matrix_device(int *m, int width) 
{ 
    int tx=blockIdx.x; 
    int ty=blockIdx.y; 
    
    int value=(tx+1)*(ty+1); 
    m[tx*width+ty] = value; 
}
__global__ void matrix_mult_device(int *Ma, int *Mb, int *Mc, int width)
{   
    
	int tx = blockIdx.x;
	int ty = blockIdx.y;
	int posfil = ty*width ,poscol = tx;

	for(int i = 0 ; i < width ; ++i)
		Mc[poscol+posfil] += Ma[posfil+i]*Mb[poscol+i*width];


	
}
void fill_matrix_host(int *m, int width) 
{ 
    for(int x=0;x<width;++x) { 
        for(int y=0;y<width;++y) { 
            int value=(x+1)*(y+1); 
            m[x*width+y] = value; 
        } 
    } 
} 

int main(void) 
{ 
    int width=2; 
    int size=width*width*sizeof(int); 

    int *m , *m1 ,*m2,*mhost; 
    m = (int *)malloc(size); 
 	m1 = (int *)malloc(size);
    m2 = (int *)malloc(size);
 	mhost = (int *)malloc(size);
 	memset(mhost, 0, size);
 	
 	
    fill_matrix_host(m, width);
    fill_matrix_host(m1, width);
    fill_matrix_host(m2, width);
   
    //hacemos la multiplicación de matrices
    for(int i = 0 ; i < width; ++i)
    	for(int j = 0 ; j < width; ++j)
    		for(int k =0 ; k < width ; ++k)
    			mhost[width*i+j] += m1[i*width + k] * m2[j + width*k];
    
    
    int *dev_m,*dev_m1,*dev_m2,*dev_mresult; 
    hipMalloc((void **)&dev_m, size); 
    hipMalloc((void **)&dev_m1, size);
    hipMalloc((void **)&dev_m2, size);
    hipMalloc((void **)&dev_mresult, size);
    dim3 dimGrid(width, width); 
    dim3 dimBlock(1, 1); 
    hipMemcpy(dev_m1, m1, size, hipMemcpyHostToDevice); 
    hipMemcpy(dev_m2, m2, size, hipMemcpyHostToDevice); 
    hipMemset(dev_mresult,0,size);
    
    fill_matrix_device<<<dimGrid, dimBlock>>>(dev_m, width); 
    matrix_mult_device<<<dimGrid, dimBlock>>>(dev_m1,dev_m2,dev_mresult, width);
    int *mok; 
    mok = (int *)malloc(size); 
    
    hipMemcpy(mok, dev_m, size, hipMemcpyDeviceToHost); //Ejercicio 5
    hipMemcpy(m2, dev_mresult, size, hipMemcpyDeviceToHost); //ahora m2 tiene el resultado del device

    int ok=1; 
    for(int i=0;i<(width*width);++i) { 
        if(m[i]!=mok[i]) ok=0; 
    } 
    
    fprintf(stdout, "%s\n", ok?"ok":"error"); //printf sobre el ejercicio 5
    
    
    //comprobar si la multiplicación a sido correcta
    ok = 1;
    for(int i = 0 ; i < (width*width);++i){
    	if(m2[i] != mhost[i]){
    		ok = 0;
			break;
    	}
    }
    fprintf(stdout, "%s\n", ok?"ok multiplicacion":"error la multiplicacion ha fallado"); 
    
    free(m); 
    free(m1);
    free(mok);
    free(m2);
    free(mhost); 
    hipFree(dev_m); 
    hipFree(dev_m1);
    hipFree(dev_m2);
    
    return 0; 
}



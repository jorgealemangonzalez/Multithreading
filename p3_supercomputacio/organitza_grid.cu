
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#define DEBUGG 1

//static const int N = 16;  //Siempre matrices cuadradas
static const int N = 36;    //Siempre matrices cuadradas
//...


//Kernel que distribueix la l'execució a la grid
__global__ void organitza_grid(int *array) {



    //Distribueix la grid(blocks i threads) com a un array unidimensional i calcula l'index d'aquesta distribució. 
    //On cada index correspon a un thread de la grid

    int idx = threadIdx.x;
    int idy = threadIdx.y;
    int idblocy = blockIdx.y;
    int idblocx = blockIdx.x;
    int width = gridDim.x * blockDim.x;
    int id_array = (idy*width + idx) + (idblocx * blockDim.x) + (idblocy * width * blockDim.y); 
    array[id_array]=(2*idblocy)+idblocx;
    
    //....
     //Recupera l'index del block a la grid
    //...

    //Guarda resultad al array

    //...

}


__host__ void printa(int *array,int sizex,int sizey)
{

//Els vostre codi...
    for(int i = 0 ; i < sizey ; ++i){      //impresion de la grid dependiendo del tamaño en el eje x e y
        for(int j = 0 ; j < sizex; ++j){
            printf("%d ",array[i*sizex+j]);
        }
        printf("\n");
    }
}   



int main(void) {

    int *dev_a  , gridsizex,gridsizey;
    int *array;
    int size = N*sizeof(int);

    // Reserva memoria al host i al device
    array = (int *)malloc(size); 

    hipMalloc((void **)&dev_a, size); 

    memset(array,0,N); //inicializamos en 0 el array

    hipMemcpy(dev_a,array,size,hipMemcpyHostToDevice); //copiamos el array del host al device

    //Crea blocks de dos dimensions amb diferent nombre de threads. Ex: Comença amb 4x4
    dim3 block_dim(sqrt(N)/2,sqrt(N)/2); //4 threads x bloque, dimension 2*2
    //...

    dim3 grid_dim(sqrt(N)/block_dim.x,sqrt(N)/block_dim.y); //numero de bloques que tendremos

    // Crea i inicialitza una grid en 2 dimensions
    //dim3 grid_dim(grid_dim,block_dim);  //la grid siempre tendra dos bloques en el eje x

    gridsizex = grid_dim.x*block_dim.x;
    gridsizey = grid_dim.y*block_dim.y;
    //...
#if DEBUGG
    printf("Dim block (x,y) %d-%d",block_dim.x,block_dim.y);
    printf("\nDim Grid (blocks)(x,y) %d-%d",grid_dim.x,grid_dim.y);
    printf("\ngrid size (threads)(x,y) %d-%d\n",gridsizex,gridsizey);
#endif

    organitza_grid<<<grid_dim, block_dim>>>(dev_a);
    hipMemcpy(array,dev_a,size,hipMemcpyDeviceToHost);


    // Printa els resultats de l'organització de la grid
    printa(array,gridsizex,gridsizey);
   



    return 0;
}
